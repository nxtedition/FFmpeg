
#include <hip/hip_runtime.h>
__device__ inline unsigned int reduce_warp(unsigned int val)
{
  for (int offset = 16; offset > 0; offset /= 2)
    val += __shfl_down_sync(0xffffffff, val, offset);

  return val;
}

__device__ inline unsigned int sumdiff_uint8(unsigned int a, unsigned int b)
{
  unsigned int sum = __vabsdiffu4(a, b);
  return (sum & 0x000000FF) + ((sum & 0x0000FF00) >> 8) + ((sum & 0x00FF0000) >> 16) + ((sum & 0xFF000000) >> 24);
}
__device__ inline unsigned int sumdiff_uint16(unsigned int a, unsigned int b)
{
  unsigned int sum = __vabsdiffu2(a, b);
  return (sum & 0x0000FFFF) + ((sum & 0xFFFF0000) >> 16);
}

__device__ void reduce(unsigned int *sums, unsigned int *sdata, hipTextureObject_t src1, hipTextureObject_t src2, unsigned int (*sumdiff)(unsigned int, unsigned int))
{
  float col_index = blockIdx.x * blockDim.x + threadIdx.x;
  float row_index = blockIdx.y * blockDim.y + threadIdx.y;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int bid = blockIdx.y * gridDim.x + blockIdx.x;

  // reads eight bytes from each source (four shorts)
  ushort4 data1 = tex2D<ushort4>(src1, col_index, row_index); 
  ushort4 data2 = tex2D<ushort4>(src2, col_index, row_index); 
  // treat the data as u32 so that we can perform vector operations
  unsigned int *data1_ptr = reinterpret_cast<unsigned int *>(&data1);
  unsigned int *data2_ptr = reinterpret_cast<unsigned int *>(&data2);
  
  unsigned int sad = sumdiff(data1_ptr[0], data2_ptr[0]); //compare the first 4 bytes (or 2 shorts)
  sad += sumdiff(data1_ptr[1], data2_ptr[1]); //compare the last 4 bytes (or 2 shorts)

  __syncthreads();
  unsigned int sum = reduce_warp(sad);

  if ((tid % 32) == 0)
    sdata[tid / 32] = sum;

  __syncthreads();

  // reduce the entire block in the first warp
  if (tid < 32)
  {
    unsigned int val = (tid < (blockDim.x * blockDim.y / 32)) ? sdata[tid] : 0;
    sum = reduce_warp(val);

    // write result for this block to global mem
    if (tid == 0)
      sums[bid] = sum;
  }
}

extern "C"
{
    __global__ void scdet_8(unsigned int *sums, hipTextureObject_t src1, hipTextureObject_t src2)
    {
        extern __shared__ unsigned int sdata[];
        reduce(sums, sdata, src1, src2, sumdiff_uint8);
    }

    __global__ void scdet_16(unsigned int *sums, hipTextureObject_t src1, hipTextureObject_t src2)
    {
        extern __shared__ unsigned int sdata[];
        reduce(sums, sdata, src1, src2, sumdiff_uint16);
    }
}

#include <hip/hip_runtime.h>
__device__ inline unsigned int reduce_warp(unsigned int val)
{
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xffffffff, val, offset);

    return val;
}

__device__ inline int count_uint8(unsigned int data, unsigned int packed_limit)
{
    unsigned int cmp = __vadd4(__vcmpgeu4(data, packed_limit), 0x01010101);
    return __popc(cmp);
}
__device__ inline int count_uint16(unsigned int data, unsigned int packed_limit)
{
    unsigned int cmp = __vadd2(__vcmpgeu2(data, packed_limit), 0x00010001);
    return __popc(cmp);
}

__device__ void reduce(unsigned int *sums, unsigned int *sdata, unsigned int packed_limit, unsigned int gridDim_x, hipTextureObject_t texObj, int (*counter)(unsigned int, unsigned int))
{
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int bid = blockIdx.y * gridDim_x + blockIdx.x;
    float col_index = blockIdx.x * blockDim.x + threadIdx.x;
    float row_index = blockIdx.y * blockDim.y + threadIdx.y;

    ushort4 data = tex2D<ushort4>(texObj, col_index, row_index);
    unsigned int *data_ptr = reinterpret_cast<unsigned int *>(&data);
    unsigned int count = counter(data_ptr[0], packed_limit);
    count += counter(data_ptr[1], packed_limit);

    __syncthreads();
    unsigned int sum = reduce_warp(count);

    if ((tid % 32) == 0)
        sdata[tid / 32] = sum;

    __syncthreads();

    // reduce the entire block in the first warp
    if (tid < 32)
    {
        unsigned int val = (tid < (blockDim.x * blockDim.y / 32)) ? sdata[tid] : 0;
        sum = reduce_warp(val);

        // write result for this block to global mem
        if (tid == 0)
            sums[bid] = sum;
    }
}

extern "C" {
__global__ void blackdetect_8(unsigned int *sums, unsigned int packed_limit, unsigned int gridDim_x, hipTextureObject_t texObj)
{
    extern __attribute__((shared)) unsigned int sdata[];
    reduce(sums, sdata, packed_limit, gridDim_x, texObj, count_uint8);
}

__global__ void blackdetect_16(unsigned int *sums, unsigned int packed_limit, unsigned int gridDim_x, hipTextureObject_t texObj)
{
    extern __attribute__((shared)) unsigned int sdata[];
    reduce(sums, sdata, packed_limit, gridDim_x, texObj, count_uint16);
}
}